#include "hip/hip_runtime.h"
#include "assignmentHPC1.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void reduce(float *input, float *output, unsigned int n) {

    unsigned int block_size = blockDim.x;
    unsigned int thread_id = threadIdx.x;
    unsigned int block_id = blockIdx.x;

    unsigned int block_start = block_id * block_size * 2 + thread_id;

    for (unsigned int stride = block_size; stride > 0; stride /= 2) {
        
        if (thread_id < stride && block_start + stride < n) {
        
            input[block_start] += input[block_start + stride];
        }
        
        __syncthreads();
    }

    if (!thread_id) {
        output[block_id] = input[block_start];
    }
}


double find_sum_cpu(double *arr_host, unsigned int N) {

    double result = 0.0f;

    cout<<"N : "<<N<<endl;

    for(unsigned int i = 0; i < N; i++) {
        result += arr_host[i];
    }

    return result;
}


void find_sum(double *arr_host, unsigned int N) {

    // ----------------------------------------- CPU Code -------------------------------------------------
    
    auto startCPU = high_resolution_clock::now();
    double result = find_sum_cpu(arr_host, N);
    auto stopCPU = high_resolution_clock::now();

    cout<<"\n\n--------------- CPU ---------------\n"<<endl;
    cout<<"Answer CPU : "<<result<<endl;
    cout<<"\nTime on CPU : "<<duration_cast<microseconds>(stopCPU - startCPU).count()/1000<<" milli seconds\n\n"<<endl;


    // ----------------------------------------- GPU Code -------------------------------------------------

}