#include "hip/hip_runtime.h"
#include "assignmentHPC1.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>

using namespace std;
using namespace std::chrono;


double find_min_cpu(double *arr_host, unsigned int N) {

    double result = numeric_limits<int>::max();

    for(unsigned int i = 0; i < N; i++) {
        if(arr_host[i] < result) {
            result = arr_host[i];
        }
    }

    return result;
}


void find_min(double *arr_host, unsigned int N) {
    
    
    // ----------------------------------------- CPU Code -------------------------------------------------

    auto startCPU = high_resolution_clock::now();
    double result = find_min_cpu(arr_host, N);
    auto stopCPU = high_resolution_clock::now();

    cout<<"\n\n--------------- CPU ---------------\n"<<endl;
    cout<<"Answer CPU : "<<result<<endl;
    cout<<"\nTime on CPU : "<<duration_cast<microseconds>(stopCPU - startCPU).count()/1000<<" milli seconds\n\n"<<endl;


    // ----------------------------------------- GPU Code -------------------------------------------------

    
}