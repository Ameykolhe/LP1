#include "hip/hip_runtime.h"
#include "assignmentHPC1.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>
#include <limits>

using namespace std;
using namespace std::chrono;



double find_max_cpu(double *arr_host, unsigned int N) {

    double result = numeric_limits<double>::min();

    for(unsigned int i = 0; i < N; i++) {
        if(arr_host[i] > result) {
            result = arr_host[i];
        }
    }

    return result;
}


void find_max(double *arr_host, unsigned int N) {


    // ----------------------------------------- CPU Code -------------------------------------------------

    auto startCPU = high_resolution_clock::now();
    double result = find_max_cpu(arr_host, N);
    auto stopCPU = high_resolution_clock::now();

    cout<<"\n\n--------------- CPU ---------------\n"<<endl;
    cout<<"Answer CPU : "<<result<<endl;
    cout<<"\nTime on CPU : "<<duration_cast<microseconds>(stopCPU - startCPU).count()/1000<<" milli seconds\n\n"<<endl;


    // ----------------------------------------- GPU Code -------------------------------------------------


}