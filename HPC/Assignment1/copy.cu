#include "hip/hip_runtime.h"
#include "copy.cuh"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <float.h>

using namespace std;


__global__ void find_minimum_kernel(float *array, float *min, int *mutex, unsigned int N,float *min_arr)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

    __shared__ float cache[256];
	
	/*
	for(int i=0; i<N; i++)
    	cache[i] = FLT_MAX;
	//__syncthreads();
	*/

	float temp = FLT_MAX;
	while(index + offset < N){
		temp = fminf(temp, array[index + offset]);
		min_arr[index]=temp;
		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fminf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*min = fminf(*min, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


void find_min_gpu(float* h_array, unsigned int N){
	cout<<"\n\n************** Computing Min using GPU *****************\n\n";

	float *d_array;
	float *h_min;
	float *d_min;
	float *min_arr;
	int *d_mutex;

    float *h_min_arr;

	// allocate memory
	h_min = (float*)malloc(sizeof(float));
    h_min_arr = (float*)malloc(N*sizeof(float));

	hipMalloc((void**)&d_array, N*sizeof(float));
    hipMalloc((void**)&d_min, sizeof(float));
    hipMalloc((void**)&min_arr, N*sizeof(float));
	hipMalloc((void**)&d_mutex, sizeof(int));

	// initialize values
	hipMemset(d_min, 0, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));

	// copy from host to device
	hipMemcpy(d_array, h_array, N*sizeof(float), hipMemcpyHostToDevice);

	// set up timing variables
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);


	// call kernel
	hipEventRecord(gpu_start, 0);

	dim3 gridSize = 256;
	dim3 blockSize = 256;

	find_minimum_kernel<<< gridSize, blockSize >>>(d_array, d_min, d_mutex, N,min_arr);

	// copy from device to host
	hipMemcpy(h_min, d_min, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_min_arr, min_arr, N*sizeof(float), hipMemcpyDeviceToHost);


	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);

	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	//report results
	cout<<"Minimum number found on gpu was: "<<*h_min<<endl;
	cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<endl;

    for(int i=0;i<N;i++)
    {
        cout<<min_arr[i]<<" ";
    }
	// free memory
    free(h_min);
    free(h_min_arr);

	hipFree(d_array);
	hipFree(d_min);
    hipFree(d_mutex);
    hipFree(min_arr);
    
    
	
}


void find_min_cpu(float* h_array, unsigned int N){
	cout<<"\n\n************** Computing Min using CPU *****************\n\n";

	float* h_min = (float*)malloc(sizeof(float));


	clock_t cpu_start = clock();

	*h_min = 3.40282e+38f;
	for(unsigned int i=0;i<N;i++){
		if(h_array[i] < *h_min){
			*h_min = h_array[i];
		}
	}

	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;


	cout<<"Minimum number found on cpu was: "<<*h_min<<endl;
	cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds\n"<<endl;

	// free memory
	free(h_min);
}
