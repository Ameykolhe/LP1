#include "hip/hip_runtime.h"
#include "findmax.cuh"

#include<float.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <limits>

using namespace std;


__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int N)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = FLT_MIN;
	while(index + offset < N){
		temp = fmaxf(temp, array[index + offset]);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


void find_max_gpu(float* h_array, unsigned int N){
	cout<<"\n\n************** Computing Max using GPU *****************\n\n";

	float *d_array;
	float *h_max;
	float *d_max;
	int *d_mutex;


	// allocate memory
	h_max = (float*)malloc(sizeof(float));

	hipMalloc((void**)&d_array, N*sizeof(float));
	hipMalloc((void**)&d_max, sizeof(float));
	hipMalloc((void**)&d_mutex, sizeof(int));

	// initialize values
	hipMemset(d_max, 0, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));

	// copy from host to device
	hipMemcpy(d_array, h_array, N*sizeof(float), hipMemcpyHostToDevice);

	// set up timing variables
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);


	// call kernel
	hipEventRecord(gpu_start, 0);

	dim3 gridSize = 256;
	dim3 blockSize = 256;

	find_maximum_kernel<<< gridSize, blockSize >>>(d_array, d_max, d_mutex, N);

	// copy from device to host
	hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);


	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);

	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	//report results
	cout<<"Maximum number found on gpu was: "<<*h_max<<endl;
	cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<endl;

	// free memory
	free(h_max);

	hipFree(d_array);
	hipFree(d_max);
	hipFree(d_mutex);
	
}


void find_max_cpu(float* h_array, unsigned int N){
	cout<<"\n\n************** Computing Max using CPU *****************\n\n";

	float* h_max = (float*)malloc(sizeof(float));


	clock_t cpu_start = clock();

	*h_max = 1.17549e-38;
	for(unsigned int i=0;i<N;i++){
		if(h_array[i] > *h_max){
			*h_max = h_array[i];
		}
	}

	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;


	cout<<"Maximum number found on cpu was: "<<*h_max<<endl;
	cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds\n"<<endl;

	// free memory
	free(h_max);
}
