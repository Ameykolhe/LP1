#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define SIZE 1024
__global__ void min(int *a, int *d)
{
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = a[i];

    __syncthreads();
    for(unsigned int s=blockDim.x/2 ; s >= 1 ; s=s/2)
    {
        if(tid < s)
        {
            if(sdata[tid] > sdata[tid + s])
            {
                sdata[tid] = sdata[tid + s];
            }
        }
    __syncthreads();
    }
    if(tid == 0 ) d[blockIdx.x] = sdata[0];
}
////////////////////////////////////////////
__global__ void max(int *a, int *c)
{
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = a[i];

    __syncthreads();

    for(unsigned int s=blockDim.x/2 ; s > 0 ; s>>=1)
    {
        if(tid < s && sdata[tid] < sdata[tid + s])
        {
            sdata[tid] = sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0 ) c[blockIdx.x] = sdata[0];
}
/////////////////////////////////////////

int main()
{
    int i;
    srand(time(NULL));

    int *a;
    a = (int*)malloc(SIZE * sizeof(int));
    int d;
    //int c;

    int *dev_a, *dev_c,*dev_d;

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));
    hipMalloc((void **) &dev_d, SIZE*sizeof(int));

    for( i = 0 ; i < SIZE ; i++)
    {
        a[i] = rand()% 1024 + 1;

    }
    for( i = 0 ; i < SIZE ; i++)
    {
        //printf("%d ",a[i]);

    }
    printf("");

    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
    //max(dev_a,dev_c);
    min<<<1,1024>>>(dev_a,dev_d);

    //cudaMemcpy(&c, dev_c, SIZE*sizeof(int),cudaMemcpyDeviceToHost);
    hipMemcpy(&d, dev_d, SIZE*sizeof(int),hipMemcpyDeviceToHost);

    //printf("max  =  %d ",c);
    printf("min  =  %d ",d);

    hipFree(dev_a);
    //cudaFree(dev_c);
    hipFree(dev_d);

    printf("");

    return 0;
    }