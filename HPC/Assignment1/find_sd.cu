#include "hip/hip_runtime.h"
#include "assignmentHPC1.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>
#include <cmath>

using namespace std;
using namespace std::chrono;


double find_sd_cpu(double *arr_host, unsigned int N) {

    double mean = 0.0f;

    for(unsigned int i = 0; i < N; i++) {
        mean += arr_host[i];
    }

    mean /= N;

    double result = 0.0f;

    for(unsigned int i = 0; i < N; i++) {
        result += pow(arr_host[i] - mean, 2);
    }

    result = sqrt(result/N);

    return result;
}


void find_sd(double *arr_host, unsigned int N) {

    // ----------------------------------------- CPU Code -------------------------------------------------

    auto startCPU = high_resolution_clock::now();
    double result = find_sd_cpu(arr_host, N);
    auto stopCPU = high_resolution_clock::now();

    cout<<"\n\n--------------- CPU ---------------\n"<<endl;
    cout<<"Answer CPU : "<<result<<endl;
    cout<<"\nTime on CPU : "<<duration_cast<microseconds>(stopCPU - startCPU).count()/1000<<" milli seconds\n\n"<<endl;
}