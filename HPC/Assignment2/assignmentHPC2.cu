#include "assignmentHPC2.cuh"

#include <iostream>

using namespace std;


int main() {

    cout<<"\n\n--------------------------- RESULTS -------------------------------\n"<<endl;

    // Vector Addition on CPU & GPU
    cout<<"\n\n--------------------------- VECT ADD\n\n"<<endl;
    vec_add();


    // Matrix Vector Multiplication on CPU & GPU
    cout<<"\n\n--------------------------- MAT VECT MUL\n\n"<<endl;
    mat_vec_mul();


    // Matrix Multiplication
    cout<<"\n\n--------------------------- MAT MUL\n\n"<<endl;
    mat_mul();

    return 0;

}
