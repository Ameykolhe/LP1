#include "hip/hip_runtime.h"
#include "assignmentHPC2.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define VEC_SIZE 1024*1024*256


__global__ void vec_add_kernel(float *a, float *b, float *c) {

        unsigned int index = blockIdx.x + threadIdx.x;
        
        if(index < VEC_SIZE) {
            c[index] = a[index] + b[index];
        }

}


void vec_add_cpu(float *a, float *b, float *c) {
    for(unsigned int i = 0; i < VEC_SIZE; i++) {
        c[i] = a[i] + b[i];
    }
}


void vec_add() {


    // declare variables
    float *a_host, *b_host, *c_host;
    float *a_device, *b_device, *c_device;

    // allocate memory to host variables
    a_host = (float *)malloc(VEC_SIZE * sizeof(float));
    b_host = (float *)malloc(VEC_SIZE * sizeof(float));
    c_host = (float *)malloc(VEC_SIZE * sizeof(float));

    // initialize host variables
    for(int i = 0; i < VEC_SIZE; i++) {
        a_host[i] = 1.0f ;//1024*1024 * float(rand())/RAND_MAX;
        b_host[i] = 1.0f ;//1024*1024 * float(rand())/RAND_MAX;
    }
    

    cout<<"INPUT SIZE "<<endl;
    cout<<"Vector A : "<<VEC_SIZE<<" * "<<1<<endl;


    // ----------------------------------------- CPU Code -------------------------------------------------
    

    // call vec_add_cpu function
    auto startCPU = high_resolution_clock::now();
    vec_add_cpu(a_host, b_host, c_host);
    auto stopCPU = high_resolution_clock::now();

    // Display Results
    cout<<"\n\n--------------- CPU ---------------\n"<<endl;
    cout<<"Answer CPU : \n"<<endl;
    for(int i = 0; i < 5; i++) {
        cout<<a_host[i]<<" + "<<b_host[i]<<" = "<<c_host[i]<<endl;
    }
    cout<<"\nTime on CPU : "<<duration_cast<microseconds>(stopCPU - startCPU).count()/1000<<" milli seconds\n\n"<<endl;

    free(c_host);




    // ----------------------------------------- GPU Code -------------------------------------------------


    // allocate memory to device vairables
    hipMalloc(&a_device, VEC_SIZE * sizeof(float));
    hipMalloc(&b_device, VEC_SIZE * sizeof(float));
    hipMalloc(&c_device, VEC_SIZE * sizeof(float));

    // copy data from host to device
    hipMemcpy(a_device, a_host, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // set up timing variables
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);


	// call kernel
	hipEventRecord(gpu_start, 0);

    // call Kernel
    dim3 blockSize(1024);
    dim3 gridSize(1024);

    auto startGPU = high_resolution_clock::now();
    vec_add_kernel<<<gridSize, blockSize>>>(a_device, b_device, c_device);

    hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);

	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

    // copy results from device to host
    c_host = (float *)malloc(VEC_SIZE * sizeof(float));
    hipMemcpy(c_host, c_device, VEC_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Display Results
    cout<<"--------------- GPU ---------------\n"<<endl;
    cout<<"Answer GPU : \n"<<endl;
    for(int i = 0; i < 5; i++) {
        cout<<a_host[i]<<" + "<<b_host[i]<<" = "<<c_host[i]<<endl;
    }
    cout<<"\nTime on GPU : "<<gpu_elapsed_time<<" milli seconds\n\n"<<endl;

    // Free allocated Memory
    free(a_host);
    free(b_host);
    free(c_host);
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

}
